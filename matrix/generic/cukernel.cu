
#include <hip/hip_runtime.h>
#ifdef NERV_GENERIC_CUKERNEL
#include <assert.h>
#include <stdio.h>
#include "matrix.h"
#include "cuda.h"
#define CUDA_THREADS_N 16
#define CUDA_THREADS_NN (16 * 16)
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))
__global__ void cudak_(sigmoid)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = 1.0 / (1.0 + exp(-a[idx]));
}

__global__ void cudak_(softmax_final)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        const MATRIX_ELEM *max, const MATRIX_ELEM *deno,
                        int nrow, int ncol, int stride, int mstride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = exp(a[idx] - max[0 + i * mstride]) / deno[0 + i * mstride];
}

__global__ void cudak_(block_reduce_sum)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_softmax_sum)(const MATRIX_ELEM *input,
                                        MATRIX_ELEM *output,
                                        const MATRIX_ELEM *max,
                                        const int istride, const int ostride,
                                        const int mstride, const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? exp(input[j + istride * blockIdx.y] - \
                                    max[0 + mstride * blockIdx.y]) : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_max)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            MATRIX_ELEM l = cudak_(arr)[threadIdx.x],
                  r = cudak_(arr)[threadIdx.x + offset];
            if (r > l) cudak_(arr)[threadIdx.x] = r;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

extern "C" {
#include "../cukernel.h"
    void cudak_(cuda_sigmoid)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(sigmoid)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b), b->nrow, b->ncol,
            b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_colsum)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_sum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_sum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_softmax_final)(const Matrix *a, const Matrix *max,
                            const Matrix *deno, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(softmax_final)<<<numBlocks, threadsPerBlock>>> \
                (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
                MATRIX_ELEM_PTR(max), MATRIX_ELEM_PTR(deno),
                b->nrow, b->ncol,
                b->stride / sizeof(MATRIX_ELEM),
                max->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_softmax_denominator)(const Matrix *a, const Matrix *max, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        assert(max->ncol == 1);
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_softmax_sum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res, MATRIX_ELEM_PTR(max),
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             max->stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_sum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_colmax)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_max)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_max)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }
}
#endif