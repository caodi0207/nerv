
#include <hip/hip_runtime.h>
#ifdef NERV_GENERIC_CUKERNEL
#include <assert.h>
#include <stdio.h>
#include "matrix.h"
#include "cuda.h"
#define CUDA_THREADS_N 16
#define CUDA_THREADS_NN (16 * 16)
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))
__global__ void cudak_(sigmoid)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = 1.0 / (1.0 + exp(-a[idx]));
}

__global__ void cudak_(softmax_final)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        const MATRIX_ELEM *max, const MATRIX_ELEM *deno,
                        int nrow, int ncol, int stride, int mstride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = exp(a[idx] - max[0 + i * mstride]) / deno[0 + i * mstride];
}

__global__ void cudak_(block_reduce_rowsum)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_colsum)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    cudak_(arr)[threadIdx.y] = i < n ? input[blockIdx.x + istride * i] : 0;
    __syncthreads();
    for (int offset = blockDim.y >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.y < offset)
            cudak_(arr)[threadIdx.y] += cudak_(arr)[threadIdx.y + offset];
        __syncthreads();
    }
    if (threadIdx.y == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_softmax_rowsum)(const MATRIX_ELEM *input,
                                        MATRIX_ELEM *output,
                                        const MATRIX_ELEM *max,
                                        const int istride, const int ostride,
                                        const int mstride, const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? exp(input[j + istride * blockIdx.y] - \
                                    max[0 + mstride * blockIdx.y]) : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_rowmax)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            MATRIX_ELEM l = cudak_(arr)[threadIdx.x],
                  r = cudak_(arr)[threadIdx.x + offset];
            if (r > l) cudak_(arr)[threadIdx.x] = r;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(add_row)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                                int nrow, int ncol, int stride, double beta) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    b[j + i * stride] += beta * a[j];
}


extern "C" {
#include "../cukernel.h"
    void cudak_(cuda_sigmoid)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(sigmoid)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b), b->nrow, b->ncol,
            b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_rowsum)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_rowsum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowsum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_colsum)(const Matrix *a, Matrix *b) {
        dim3 block(1, CUDA_THREADS_NN);
        int nrow = a->nrow;
        int blocks_per_col = CEIL_DIV(nrow, block.x);
        dim3 grid(a->ncol, blocks_per_col);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, a->ncol * sizeof(MATRIX_ELEM), blocks_per_col);
        cudak_(block_reduce_colsum)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             nrow);
        nrow = blocks_per_col;
        assert((unsigned long)nrow <= block.y);
        grid.y = 1;
        cudak_(block_reduce_colsum)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             nrow);
        cudaFree(res);
    }

    void cudak_(cuda_softmax_final)(const Matrix *a, const Matrix *max,
                            const Matrix *deno, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(softmax_final)<<<numBlocks, threadsPerBlock>>> \
                (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
                MATRIX_ELEM_PTR(max), MATRIX_ELEM_PTR(deno),
                b->nrow, b->ncol,
                b->stride / sizeof(MATRIX_ELEM),
                max->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_softmax_denominator)(const Matrix *a, const Matrix *max, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        assert(max->ncol == 1);
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_softmax_rowsum) \
            <<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res, MATRIX_ELEM_PTR(max),
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             max->stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowsum) \
            <<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_rowmax)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_rowmax)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowmax)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    /* in-place calc */
    void cudak_(cuda_add_row)(const Matrix *a, Matrix *b, double beta) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(add_row)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b), b->nrow, b->ncol,
            b->stride / sizeof(MATRIX_ELEM), beta);
    }
}
#endif
