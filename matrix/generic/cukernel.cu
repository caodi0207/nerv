
#include <hip/hip_runtime.h>
#ifdef NERV_GENERIC_CUKERNEL
#include <assert.h>
#include <stdio.h>
#include "matrix.h"
#include "cuda.h"
#include "float.h"
#define CUDA_THREADS_N 16
#define CUDA_THREADS_NN ((CUDA_THREADS_N) * (CUDA_THREADS_N))
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))
__global__ void cudak_(log_elem)(const MATRIX_ELEM *a, MATRIX_ELEM *b, 
                                int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    MATRIX_ELEM tmp;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    tmp = a[idx];
    if(tmp < FLT_MIN) tmp = FLT_MIN;
    b[idx] = log(tmp);
}

__global__ void cudak_(mul_elem)(const MATRIX_ELEM *a, const MATRIX_ELEM *b,
                                MATRIX_ELEM *c, 
                                int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    c[idx] = a[idx] * b[idx];
}

__global__ void cudak_(sigmoid)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = 1.0 / (1.0 + exp(-a[idx]));
}

__global__ void cudak_(sigmoid_grad)(const MATRIX_ELEM *output,
                                    const MATRIX_ELEM *err,
                                    MATRIX_ELEM *nerr,
                                    int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    nerr[idx] = output[idx] * (1.0 - output[idx]) * err[idx];
}

__global__ void cudak_(softmax_final)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        const MATRIX_ELEM *max, const MATRIX_ELEM *deno,
                        int nrow, int ncol, int stride, int mstride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = exp(a[idx] - max[0 + i * mstride]) / deno[0 + i * mstride];
}

__global__ void cudak_(block_reduce_rowsum)(const MATRIX_ELEM *input,
                                            MATRIX_ELEM *output,
                                            const int istride, const int ostride,
                                            const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_colsum)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    cudak_(arr)[threadIdx.y] = i < n ? input[blockIdx.x + istride * i] : 0;
    __syncthreads();
    for (int offset = blockDim.y >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.y < offset)
            cudak_(arr)[threadIdx.y] += cudak_(arr)[threadIdx.y + offset];
        __syncthreads();
    }
    if (threadIdx.y == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_colsame)(const MATRIX_ELEM *input,
                                            const MATRIX_ELEM *ref_input,
                                            MATRIX_ELEM *output,
                                            const int istride, const int ostride,
                                            const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    cudak_(arr)[threadIdx.y] = (i < n && input[blockIdx.x + istride * i] == \
                                        ref_input[blockIdx.x + istride * i]) ? 1.0 : 0;
    __syncthreads();
    for (int offset = blockDim.y >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.y < offset)
            cudak_(arr)[threadIdx.y] += cudak_(arr)[threadIdx.y + offset];
        __syncthreads();
    }
    if (threadIdx.y == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_softmax_rowsum)(const MATRIX_ELEM *input,
                                        MATRIX_ELEM *output,
                                        const MATRIX_ELEM *max,
                                        const int istride, const int ostride,
                                        const int mstride, const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? exp(input[j + istride * blockIdx.y] - \
                                    max[0 + mstride * blockIdx.y]) : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_rowmax)(const MATRIX_ELEM *input,
                                            MATRIX_ELEM *output,
                                            const int istride, const int ostride,
                                            const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            MATRIX_ELEM l = cudak_(arr)[threadIdx.x],
                        r = cudak_(arr)[threadIdx.x + offset];
            if (r > l)
                cudak_(arr)[threadIdx.x] = r;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_rowmax_idx)(const MATRIX_ELEM *input,
                                                const MATRIX_ELEM *idx_input,
                                                MATRIX_ELEM *output,
                                                MATRIX_ELEM *idx_output,
                                                const int istride, const int ostride,
                                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    MATRIX_ELEM *arr_val = cudak_(arr);
    MATRIX_ELEM *arr_idx = arr_val + blockDim.x;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    arr_val[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    arr_idx[threadIdx.x] = j < n ? idx_input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            MATRIX_ELEM l = arr_val[threadIdx.x],
                        r = arr_val[threadIdx.x + offset];
            if (r > l)
            {
                arr_val[threadIdx.x] = r;
                arr_idx[threadIdx.x] = arr_idx[threadIdx.x + offset];
            }
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        output[blockIdx.x + ostride * blockIdx.y] = arr_val[0];
        idx_output[blockIdx.x + ostride * blockIdx.y] = arr_idx[0];
    }
}

__global__ void cudak_(add_row)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                                int nrow, int ncol, int stride, double beta) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    b[j + i * stride] += beta * a[j];
}

__global__ void cudak_(fill)(MATRIX_ELEM *a,
                            int nrow, int ncol, int stride, double val) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    a[j + i * stride] = val;
}

__global__ void cudak_(expand_frm)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                                    int nrow, int ncol,
                                    int enrow, int encol,
                                    int stride, int estride,
                                    int context) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int ridx;
    if (i >= enrow || j >= encol) return;
    ridx = i + j / ncol - context;
    if (ridx < 0) ridx = 0;
    else if (ridx >= nrow) ridx = nrow - 1;
    b[j + i * estride] = a[j % ncol + ridx * stride];
}

__global__ void cudak_(rearrange_frm)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                                    int nrow, int ncol,
                                    int stride, int step, int orig_dim) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    b[j + i * stride] = a[j / step + (j % step) * orig_dim + i * stride];
}

__global__ void cudak_(scale_row)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                                    int nrow, int ncol,
                                    int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    b[j + i * stride] *= a[j];
}

__global__ void cudak_(decompress)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                                    int nrow, int ncol,
                                    int stride_a, int stride_b) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    b[lrintf(a[j + i * stride_a]) + i * stride_b] = 1.0;
}

__global__ void cudak_(gen_col_idx)(MATRIX_ELEM *b,
                                    int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    b[j + i * stride] = j;
}

extern "C" {
#include "../cukernel.h"
    void cudak_(cuda_log_elem)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(log_elem)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             b->nrow, b->ncol, b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_mul_elem)(const Matrix *a, const Matrix *b,
                                Matrix *c) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(mul_elem)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             MATRIX_ELEM_PTR(c),
             b->nrow, b->ncol, b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_sigmoid)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(sigmoid)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b), b->nrow, b->ncol,
            b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_sigmoid_grad)(const Matrix *output,
                                    const Matrix *err, Matrix *nerr) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(nerr->ncol, threadsPerBlock.x),
                CEIL_DIV(nerr->nrow, threadsPerBlock.y));
        cudak_(sigmoid_grad)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(output), MATRIX_ELEM_PTR(err),
             MATRIX_ELEM_PTR(nerr),
             nerr->nrow, nerr->ncol,
             nerr->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_rowsum)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_rowsum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowsum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_colsame)(const Matrix *a, const Matrix *ref, Matrix *b) {
        dim3 block(1, CUDA_THREADS_NN);
        int nrow = a->nrow;
        int blocks_per_col = CEIL_DIV(nrow, block.y);
        dim3 grid(a->ncol, blocks_per_col);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, a->ncol * sizeof(MATRIX_ELEM), blocks_per_col);
        cudak_(block_reduce_colsame)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(ref), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             nrow);
        nrow = blocks_per_col;
        assert((unsigned long)nrow <= block.y);
        grid.y = 1;
        cudak_(block_reduce_colsum)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             nrow);
        cudaFree(res);
    }

    void cudak_(cuda_colsum)(const Matrix *a, Matrix *b) {
        dim3 block(1, CUDA_THREADS_NN);
        int nrow = a->nrow;
        int blocks_per_col = CEIL_DIV(nrow, block.y);
        dim3 grid(a->ncol, blocks_per_col);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, a->ncol * sizeof(MATRIX_ELEM), blocks_per_col);
        cudak_(block_reduce_colsum)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             nrow);
        nrow = blocks_per_col;
        assert((unsigned long)nrow <= block.y);
        grid.y = 1;
        cudak_(block_reduce_colsum)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             nrow);
        cudaFree(res);
    }

    void cudak_(cuda_softmax_final)(const Matrix *a, const Matrix *max,
                            const Matrix *deno, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(softmax_final)<<<numBlocks, threadsPerBlock>>> \
                (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
                MATRIX_ELEM_PTR(max), MATRIX_ELEM_PTR(deno),
                b->nrow, b->ncol,
                b->stride / sizeof(MATRIX_ELEM),
                max->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_softmax_denominator)(const Matrix *a, const Matrix *max, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        assert(max->ncol == 1);
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_softmax_rowsum) \
            <<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res, MATRIX_ELEM_PTR(max),
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             max->stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowsum) \
            <<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_rowmax)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_rowmax)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowmax)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_rowmax_idx)(const Matrix *a, Matrix *b, Matrix *b_idx) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *a_idx, *res, *res_idx;
        size_t stride;
        cudaMallocPitch(&a_idx, &stride, a->stride, a->nrow);
        cudak_(gen_col_idx)<<<grid, block>>>(a_idx, a->nrow, ncol, stride / sizeof(MATRIX_ELEM));
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudaMallocPitch(&res_idx, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_rowmax_idx)<<<grid, block,
                                        2 * block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), a_idx, res, res_idx,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(a_idx);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowmax_idx)<<<grid, block,
                                        2 * block.x * sizeof(MATRIX_ELEM)>>> \
            (res, res_idx, MATRIX_ELEM_PTR(b), MATRIX_ELEM_PTR(b_idx),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
        cudaFree(res_idx);
    }

    /* in-place calc */
    void cudak_(cuda_add_row)(const Matrix *a, Matrix *b, double beta) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(add_row)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b), b->nrow, b->ncol,
            b->stride / sizeof(MATRIX_ELEM), beta);
    }

    void cudak_(cuda_fill)(Matrix *a, double val) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(a->ncol, threadsPerBlock.x),
                CEIL_DIV(a->nrow, threadsPerBlock.y));
        cudak_(fill)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), a->nrow, a->ncol,
            a->stride / sizeof(MATRIX_ELEM), val);
    }

    void cudak_(cuda_expand_frm)(const Matrix *a, Matrix *b, int context) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(expand_frm)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             a->nrow, a->ncol,
             b->nrow, b->ncol,
             a->stride / sizeof(MATRIX_ELEM),
             b->stride / sizeof(MATRIX_ELEM),
             context);
    }

    void cudak_(cuda_rearrange_frm)(const Matrix *a, Matrix *b, int step) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(rearrange_frm)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             b->nrow, b->ncol, b->stride / sizeof(MATRIX_ELEM),
             step, b->ncol / step);
    }

    void cudak_(cuda_scale_row)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N, CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(scale_row)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             b->nrow, b->ncol, b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_decompress)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(1, CUDA_THREADS_NN);
        dim3 numBlocks(1, CEIL_DIV(a->nrow, threadsPerBlock.y));
        cudak_(decompress)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             a->nrow, a->ncol,
             a->stride / sizeof(MATRIX_ELEM),
             b->stride / sizeof(MATRIX_ELEM));
    }
}
#endif
