
#include <hip/hip_runtime.h>
#ifdef NERV_GENERIC_CUKERNEL
#include <assert.h>
#include <stdio.h>
#include "matrix.h"
#include "cuda.h"
#define CUDA_THREADS_N 16
#define CUDA_THREADS_NN (16 * 16)
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))
__global__ void cudak_(log_elem)(const MATRIX_ELEM *a, MATRIX_ELEM *b, 
                                int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = log(a[idx]);
}

__global__ void cudak_(mul_elem)(const MATRIX_ELEM *a, const MATRIX_ELEM *b,
                                MATRIX_ELEM *c, 
                                int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    c[idx] = a[idx] * b[idx];
}

__global__ void cudak_(sigmoid)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = 1.0 / (1.0 + exp(-a[idx]));
}

__global__ void cudak_(sigmoid_grad)(const MATRIX_ELEM *output,
                                    const MATRIX_ELEM *err,
                                    MATRIX_ELEM *nerr,
                                    int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    nerr[idx] = output[idx] * (1.0 - output[idx]) * err[idx];
}

__global__ void cudak_(softmax_final)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                        const MATRIX_ELEM *max, const MATRIX_ELEM *deno,
                        int nrow, int ncol, int stride, int mstride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = exp(a[idx] - max[0 + i * mstride]) / deno[0 + i * mstride];
}

__global__ void cudak_(block_reduce_rowsum)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_colsum)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    cudak_(arr)[threadIdx.y] = i < n ? input[blockIdx.x + istride * i] : 0;
    __syncthreads();
    for (int offset = blockDim.y >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.y < offset)
            cudak_(arr)[threadIdx.y] += cudak_(arr)[threadIdx.y + offset];
        __syncthreads();
    }
    if (threadIdx.y == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_softmax_rowsum)(const MATRIX_ELEM *input,
                                        MATRIX_ELEM *output,
                                        const MATRIX_ELEM *max,
                                        const int istride, const int ostride,
                                        const int mstride, const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? exp(input[j + istride * blockIdx.y] - \
                                    max[0 + mstride * blockIdx.y]) : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            cudak_(arr)[threadIdx.x] += cudak_(arr)[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(block_reduce_rowmax)(const MATRIX_ELEM *input,
                                MATRIX_ELEM *output,
                                const int istride, const int ostride,
                                const int n) {
    extern __shared__ MATRIX_ELEM cudak_(arr)[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    cudak_(arr)[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            MATRIX_ELEM l = cudak_(arr)[threadIdx.x],
                  r = cudak_(arr)[threadIdx.x + offset];
            if (r > l) cudak_(arr)[threadIdx.x] = r;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = cudak_(arr)[0];
}

__global__ void cudak_(add_row)(const MATRIX_ELEM *a, MATRIX_ELEM *b,
                                int nrow, int ncol, int stride, double beta) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    b[j + i * stride] += beta * a[j];
}

__global__ void cudak_(fill)(MATRIX_ELEM *a,
                            int nrow, int ncol, int stride, double val) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= nrow || j >= ncol) return;
    a[j + i * stride] = val;
}


extern "C" {
#include "../cukernel.h"
    void cudak_(cuda_log_elem)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(log_elem)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             b->nrow, b->ncol, b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_mul_elem)(const Matrix *a, const Matrix *b,
                                Matrix *c) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(mul_elem)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
             MATRIX_ELEM_PTR(c),
             b->nrow, b->ncol, b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_sigmoid)(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(sigmoid)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b), b->nrow, b->ncol,
            b->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_sigmoid_grad)(const Matrix *output,
                                    const Matrix *err, Matrix *nerr) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(nerr->ncol, threadsPerBlock.x),
                CEIL_DIV(nerr->nrow, threadsPerBlock.y));
        cudak_(sigmoid_grad)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(output), MATRIX_ELEM_PTR(err),
             MATRIX_ELEM_PTR(nerr),
             nerr->nrow, nerr->ncol,
             nerr->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_rowsum)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_rowsum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowsum)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_colsum)(const Matrix *a, Matrix *b) {
        dim3 block(1, CUDA_THREADS_NN);
        int nrow = a->nrow;
        int blocks_per_col = CEIL_DIV(nrow, block.x);
        dim3 grid(a->ncol, blocks_per_col);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, a->ncol * sizeof(MATRIX_ELEM), blocks_per_col);
        cudak_(block_reduce_colsum)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             nrow);
        nrow = blocks_per_col;
        assert((unsigned long)nrow <= block.y);
        grid.y = 1;
        cudak_(block_reduce_colsum)<<<grid, block, block.y * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             nrow);
        cudaFree(res);
    }

    void cudak_(cuda_softmax_final)(const Matrix *a, const Matrix *max,
                            const Matrix *deno, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(softmax_final)<<<numBlocks, threadsPerBlock>>> \
                (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b),
                MATRIX_ELEM_PTR(max), MATRIX_ELEM_PTR(deno),
                b->nrow, b->ncol,
                b->stride / sizeof(MATRIX_ELEM),
                max->stride / sizeof(MATRIX_ELEM));
    }

    void cudak_(cuda_softmax_denominator)(const Matrix *a, const Matrix *max, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        assert(max->ncol == 1);
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_softmax_rowsum) \
            <<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res, MATRIX_ELEM_PTR(max),
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             max->stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowsum) \
            <<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    void cudak_(cuda_rowmax)(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        MATRIX_ELEM *res;
        size_t stride;
        cudaMallocPitch(&res, &stride, blocks_per_row * sizeof(MATRIX_ELEM), a->nrow);
        cudak_(block_reduce_rowmax)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (MATRIX_ELEM_PTR(a), res,
             a->stride / sizeof(MATRIX_ELEM), stride / sizeof(MATRIX_ELEM),
             ncol);
        ncol = blocks_per_row;
        assert((unsigned long)ncol <= block.x);
        grid.x = 1;
        cudak_(block_reduce_rowmax)<<<grid, block, block.x * sizeof(MATRIX_ELEM)>>> \
            (res, MATRIX_ELEM_PTR(b),
             stride / sizeof(MATRIX_ELEM), b->stride / sizeof(MATRIX_ELEM),
             ncol);
        cudaFree(res);
    }

    /* in-place calc */
    void cudak_(cuda_add_row)(const Matrix *a, Matrix *b, double beta) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        cudak_(add_row)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), MATRIX_ELEM_PTR(b), b->nrow, b->ncol,
            b->stride / sizeof(MATRIX_ELEM), beta);
    }

    void cudak_(cuda_fill)(Matrix *a, double val) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(a->ncol, threadsPerBlock.x),
                CEIL_DIV(a->nrow, threadsPerBlock.y));
        cudak_(fill)<<<numBlocks, threadsPerBlock>>> \
            (MATRIX_ELEM_PTR(a), a->nrow, a->ncol,
            a->stride / sizeof(MATRIX_ELEM), val);
    }
}
#endif
