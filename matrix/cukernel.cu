#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "generic/matrix.h"
#include "hip/hip_runtime.h"
#define CUDA_THREADS_N 16
#define CUDA_THREADS_NN (16 * 16)
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))
__global__ void sigmoid(const float *a, float *b,
                        int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = 1.0 / (1.0 + exp(-a[idx]));
}

__global__ void softmax_final(const float *a, float *b,
                        const float *max, const float *deno,
                        int nrow, int ncol, int stride, int mstride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = exp(a[idx] - max[0 + i * mstride]) / deno[0 + i * mstride];
}

__global__ void block_reduce_sum(const float *input, float *output,
                        const int istride, const int ostride,
                        const int n) {
    extern __shared__ float arr[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    arr[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            arr[threadIdx.x] += arr[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = arr[0];
}

__global__ void block_reduce_softmax_sum(const float *input, float *output,
                                        const float *max,
                                        const int istride, const int ostride,
                                        const int mstride, const int n) {
    extern __shared__ float arr[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    arr[threadIdx.x] = j < n ? exp(input[j + istride * blockIdx.y] - \
                                    max[0 + mstride * blockIdx.y]) : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
            arr[threadIdx.x] += arr[threadIdx.x + offset];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = arr[0];
}

__global__ void block_reduce_max(const float *input, float *output,
                        const int istride, const int ostride,
                        const int n) {
    extern __shared__ float arr[];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    arr[threadIdx.x] = j < n ? input[j + istride * blockIdx.y] : 0;
    __syncthreads();
    for (int offset = blockDim.x >> 1;  offset; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            float l = arr[threadIdx.x],
                  r = arr[threadIdx.x + offset];
            if (r > l) arr[threadIdx.x] = r;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        output[blockIdx.x + ostride * blockIdx.y] = arr[0];
}

extern "C" {
    void cuda_sigmoid(const Matrix *a, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        sigmoid<<<numBlocks, threadsPerBlock>>>(a->data.f, b->data.f, b->nrow, b->ncol,
                b->stride / sizeof(float));
    }

    void cuda_colsum(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        float *res;
        size_t stride;
        hipMallocPitch(&res, &stride, blocks_per_row * sizeof(float), a->nrow);
        block_reduce_sum<<<grid, block, block.x * sizeof(float)>>> \
            (a->data.f, res,
             a->stride / sizeof(float), stride / sizeof(float),
             ncol);
        ncol = blocks_per_row;
        assert(ncol <= block.x);
        grid.x = 1;
        block_reduce_sum<<<grid, block, block.x * sizeof(float)>>> \
            (res, b->data.f,
             stride / sizeof(float), b->stride / sizeof(float),
             ncol);
        hipFree(res);
    }

    void cuda_softmax_final(const Matrix *a, const Matrix *max,
                            const Matrix *deno, Matrix *b) {
        dim3 threadsPerBlock(CUDA_THREADS_N,
                CUDA_THREADS_N);
        dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                CEIL_DIV(b->nrow, threadsPerBlock.y));
        softmax_final<<<numBlocks, threadsPerBlock>>>(a->data.f, b->data.f,
                max->data.f, deno->data.f,
                b->nrow, b->ncol,
                b->stride / sizeof(float),
                max->stride / sizeof(float));
    }

    void cuda_softmax_denominator(const Matrix *a, const Matrix *max, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        float *res;
        size_t stride;
        assert(max->ncol == 1);
        hipMallocPitch(&res, &stride, blocks_per_row * sizeof(float), a->nrow);
        block_reduce_softmax_sum<<<grid, block, block.x * sizeof(float)>>> \
            (a->data.f, res, max->data.f,
             a->stride / sizeof(float), stride / sizeof(float),
             max->stride / sizeof(float),
             ncol);
        ncol = blocks_per_row;
        assert(ncol <= block.x);
        grid.x = 1;
        block_reduce_sum<<<grid, block, block.x * sizeof(float)>>> \
            (res, b->data.f,
             stride / sizeof(float), b->stride / sizeof(float),
             ncol);
        hipFree(res);
    }

    void cuda_colmax(const Matrix *a, Matrix *b) {
        dim3 block(CUDA_THREADS_NN, 1);
        int ncol = a->ncol;
        int blocks_per_row = CEIL_DIV(ncol, block.x);
        dim3 grid(blocks_per_row, a->nrow);
        float *res;
        size_t stride;
        hipMallocPitch(&res, &stride, blocks_per_row * sizeof(float), a->nrow);
        block_reduce_max<<<grid, block, block.x * sizeof(float)>>> \
            (a->data.f, res,
             a->stride / sizeof(float), stride / sizeof(float),
             ncol);
        ncol = blocks_per_row;
        assert(ncol <= block.x);
        grid.x = 1;
        block_reduce_max<<<grid, block, block.x * sizeof(float)>>> \
            (res, b->data.f,
             stride / sizeof(float), b->stride / sizeof(float),
             ncol);
        hipFree(res);
    }
}
