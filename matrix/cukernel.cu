#include "hip/hip_runtime.h"
#include "generic/matrix.h"
#define CUDA_THREADS_N 16
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))
__global__ void sigmoid(const float *a, float *b,
                        int nrow, int ncol, int stride) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    long idx;
    if (i >= nrow || j >= ncol) return;
    idx = j + i * stride;
    b[idx] = 1.0 / (1.0 + exp(-a[idx]));
}

extern "C" void cuda_sigmoid(const Matrix *a, Matrix *b) {
    dim3 threadsPerBlock(CUDA_THREADS_N,
                         CUDA_THREADS_N);
    dim3 numBlocks(CEIL_DIV(b->ncol, threadsPerBlock.x),
                    CEIL_DIV(b->nrow, threadsPerBlock.y));
    sigmoid<<<numBlocks, threadsPerBlock>>>(a->data.f, b->data.f, b->nrow, b->ncol,
                                            b->stride / sizeof(float));
}
